
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int N = (1024 * 1024);
const int BSIZE = 1024;

__global__ void gpu_kernel(float *d_A, float *d_B, float *d_C, int len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < len)
        d_C[i] = d_A[i] + d_B[i];
}

__host__ void cpu_kernel(float *d_A, float *d_B, float *d_C, int len)
{
    for (int i = 0; i < len; i++)
        d_C[i] = d_A[i] + d_B[i];
}

int main(int argc, char **argv)
{
    float *h_A, *h_B, *h_C;  // for host memory
    float *d_A, *d_B, *d_C;  // for device memory
    float result;     // resut
    dim3 grid(N/BSIZE, 1, 1), block(BSIZE, 1, 1); // grid and block size

    /* host memory allocation */
    h_A = (float *)malloc(sizeof(float) * N);
    h_B = (float *)malloc(sizeof(float) * N);
    h_C = (float *)malloc(sizeof(float) * N);

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f; h_B[i] = 2.0f; h_C[i] = 0.0f;
    }

    /* device memory allocation */
    hipMalloc((void **)&d_A, sizeof(float) * N);
    hipMalloc((void **)&d_B, sizeof(float) * N);
    hipMalloc((void **)&d_C, sizeof(float) * N);

    /* copy data the host to the device */
    hipMemcpy(d_A, h_A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * N, hipMemcpyHostToDevice);

    /* The host calles the karnel */ 
    gpu_kernel<<<grid, block>>>(d_A, d_B, d_C, N);

    /*  Result write back */
    hipMemcpy(h_C, d_C, sizeof(float) * N, hipMemcpyDeviceToHost);

    /*  Release device memory */
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    /* check the result for GPU */
    result = 0.0;
    for (int i = 0; i < N; ++i)
        result += h_C[i];
    result /= (float) N;
    printf("GPU: result = %f\n", result);

    /* check the result for CPU */
    result = 0.0;
    cpu_kernel(h_A, h_B, h_C, N);
    for (int i = 0; i < N; ++i)
        result += h_C[i];
    result /= (float) N;
    printf("CPU: result = %f\n", result);

    /*  Release host memory  */
    free(h_A); free(h_B); free(h_C);

  return 0;
}
